﻿#include "Utilities.h"

#include <iostream>


void utils::PrintMatrix(const double *mat, unsigned n, unsigned m)
{
	std::cout << "n = " << n << " m = " << m << std::endl;
	for (auto i = 0; i < static_cast<decltype(i)>(n); ++i)
	{
		for (auto j = 0; j < static_cast<decltype(j)>(m); ++j)
		{
			std::cout << mat[i * m + j] << " ";
		}
		std::cout << std::endl;
	}
}

void utils::CheckError(hipError_t error, unsigned line)
{
	if (error != hipSuccess)
	{
		std::cout << "Error in line " << line << ": " << hipGetErrorString(error) << std::endl;
		exit(1);
	}
}

void utils::CheckError(hipError_t error, unsigned line, std::function<void(void)> cleanup)
{
	if (error != hipSuccess)
	{
		std::cout << "Error in line " << line << ": " << hipGetErrorString(error) << std::endl;
		exit(1);
	}
}